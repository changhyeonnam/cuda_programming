//
// Created by changhyeonnam on 2023/01/10.
//


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

// CUDA kernel for vector addition
// __global__ means this is called from CPU, and runs on the GPU
__global__ void vectorAdd(const int *__restrict a, const int *__restrict b,
                          int *__restrict c, int N){
    // Calculate global thread ID
    // blockDim = 1 dim (just integer)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Boundary check
    if (tid<N)
        // Each thread adds a single element
        c[tid] = a[tid] + b[tid];
}

// Initialize vector of size n to int between 0~99
void matrix_init(int* a, int n){
    for(int i=0; i<n; i++){
        a[i] = rand() % 100;
    }
}
// Check vector add result
void error_check(int* a, int* b, int* c, int n){
    for(int i=0; i<n; i++){
        assert(c[i] == a[i] + b[i]);
    }
}

// print vector add result
void print_result(int* a, int* b, int* c, int n){
    for(int i=0; i<n; i++){
        if(i%100==0)
            std::cout<<"c["<<i<<"]="<<c[i]<<" = "<<"a["<<i<<"]="<<a[i]<<" + " <<"b["<<i<<"]="<<b[i]<<'\n';
    }
}


int main(){
    // Vector size of 2^16 (65536 elements)
    int n = 1<<16;

    // Host vector pointers
    int *h_a, *h_b, *h_c;

    // Device vector pointers
    int *d_a, *d_b, *d_c;

    // Allocation size for all vectors
    size_t bytes = sizeof(int) * n;

    // Allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate device(gpu) memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    /* There is something called unified memory.
     * one set of memory that gets migrated between the GPU and CPU viceversa.
     * [next lecture]
     */

    // Initialize vectors a and b with random values between 0 and 99
    matrix_init(h_a, n);
    matrix_init(h_b, n);

    // Copy data from the CPU(HOST) to the GPU
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threadblock size
    // it's generally good to do this a size of 32 because these have to translate it to warps.
    // which are of size 32.
    int NUM_THREADS = 256;

    // Grid size
    // NUM_THREAD * NUM_BLOCKS = NUMBER of Elements.
    int NUM_BLOCKS = (int)ceil(n/NUM_THREADS);

    // Launch kernel on default strem w/o
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

    // Copy sum vector from device to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Check result for errors
    error_check(h_a, h_b, h_c, n);
    print_result(h_a, h_b, h_c, n);
    printf("COMPLETED SUCCESFULLY\n");
    return 0;
}
